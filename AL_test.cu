#include "hip/hip_runtime.h"
#pragma once

/*--------------------------------------- INCLUDES ------------------------------------------------ */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <time.h>
#include <chrono>
#include <ctime>
#include <sstream>
#include <cstdlib>
#include <unistd.h>
#include <float.h>
#include <functional>
#include <cstring>
#include <cstdarg>
#include <random>
#include <unordered_set>
#include <hip/hip_runtime.h>

#include "types.h"
#include "kernels.cu"


/*--------------------------------------- MACROS -------------------------------------------------- */

#define CHECK(call)                                                                         \
	{                                                                                       \
		const hipError_t err = call;                                                       \
		if (err != hipSuccess) {                                                           \
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
			exit(EXIT_FAILURE);                                                             \
		}                                                                                   \
	}

#define CHECK_KERNELCALL()                                                                  \
	{                                                                                       \
		const hipError_t err = hipGetLastError();                                         \
		if (err != hipSuccess) {                                                           \
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
			exit(EXIT_FAILURE);                                                             \
		}                                                                                   \
	} 


/*--------------------------------------- GLOBAL VARIABLES ---------------------------------------- */

extern double MAX_MU;
extern double MAX_LAMBDA;

extern char name_suffix[20];  
extern char results_path[100];

extern bool Q_DIAG;
extern bool Q_ID;
extern bool PCR_PROBLEM;


/*--------------------------------------- ENUMS --------------------------------------------------- */ 
enum stop_conditions_names {max_Al_attempts, max_mu, max_lambda, stop_conditions_end};
enum fill_distributions {uniform, MMF, PCR, PCRL, fill_distributions_end};


/*--------------------------------------- STRUCTS ------------------------------------------------- */

struct test_results{
    dim_Type N;
    dim_Type M;
    double correct_ratio;
    double unfinished_ratio;
    double normalized_error_mean;
    float mean_al_attempts_on_correct_solutions;
    float mean_al_attempts_on_wrong_solutions;
    float mean_al_attempts_on_unfinished_solutions;
    lambda_Type mean_lambda_on_correct_solutions;
    lambda_Type mean_lambda_on_unfinished_solutions;
    lambda_Type mean_lambda_on_wrong_solutions;
    mu_Type mean_mu_on_correct_solutions;
    mu_Type mean_mu_on_unfinished_solutions;
    mu_Type mean_mu_on_wrong_solutions;
    lambda_Type lambda_min_on_correct_solutions;
    lambda_Type lambda_min_on_unfinished_solutions;
    lambda_Type lambda_min_on_wrong_solutions;
    lambda_Type lambda_max_on_correct_solutions;
    lambda_Type lambda_max_on_unfinished_solutions;
    lambda_Type lambda_max_on_wrong_solutions;
    double duration;
};


/*--------------------------------------- PARTIAL RESULT COMPUTING FUNCTIONS ---------------------- */

int test_at_dimension(  dim_Type N, dim_Type M, int MAXITER, int N_AL_ATTEMPTS, mu_Type initial_mu, lambda_Type initial_lambda,  mu_Type rho, 
                        void (*fill_Q)(Q_Type *Q, const dim_Type N, const Q_Type lowerbound_or_unused, const Q_Type upperbound_or_unused), Q_Type lb_Q, Q_Type ub_Q, 
                        void (*fill_A)(A_Type* A, const dim_Type M, const dim_Type N, const float one_probability_or_unused, const b_Type b_or_unused), float one_prob, 
                        void (*fill_b)(b_Type* b, const dim_Type M, const b_Type b_val_or_unused), b_Type b_val, 
                        std::function<bool(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c)> al_end_condition, 
                        mu_Type (*update_mu)(const mu_Type mu, const mu_Type rho), 
                        test_results* results, bool verbose, bool strong_verbose);

Q_Type compute_xQx(const Q_Type* __restrict__ Q, const bool* __restrict__ x, dim_Type N);

Q_Type compute_max(const Q_Type* __restrict__ Q, dim_Type N);


void compute_Q_plus_AT_A_upper_triangular_lin(const Q_Type* __restrict__ Q, A_Type* __restrict__ A, A_Type* __restrict__ Q_plus_AT_A, const dim_Type M, const dim_Type N);


/*--------------------------------------- FILL FUNCTIONS ------------------------------------------ */

void fill_Q_id_lin(Q_Type*  Q, const dim_Type N, const Q_Type not_used_1, const Q_Type not_used_2);
void fill_Q_diag_lin(Q_Type*  Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbund);
void fill_Q_upper_trianular_lin(Q_Type *Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbound);
void fill_Q_manual_lin(Q_Type*  Q, const dim_Type N, const Q_Type unused_1, const Q_Type unused_2) {printf("not yet implemented"); exit(0);}

void fill_A_neg_binary_lin(A_Type*  A, const dim_Type M, const dim_Type N, const float one_probability, const b_Type b);
void fill_A_manual_lin(A_Type*  A, const dim_Type M, const dim_Type N, const float unused_1, const b_Type unused_2) {printf("not yet implemented"); exit(0);}

void fill_b_vector_lin(b_Type* b, const dim_Type M, const b_Type b_val);
void fill_b_manual_lin(b_Type* b, const dim_Type M, const b_Type unused) {printf("not yet implemented"); exit(0);}

void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda);
void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda, lambda_Type noise_amplitude);


/*--------------------------------------- INDEX CONVERTION FUNCTIONS ------------------------------ */

inline unsigned int triang_index(dim_Type i, dim_Type j, dim_Type N){
    return (unsigned int)(i * (N - 0.5f) - i*i/2.0f + j);
}

/*--------------------------------------- HALT CONDITION FUNCTIONS -------------------------------- */

inline bool max_Al_attempts_condition(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c){
    return i < N_AL_ATTEMPTS;
}

inline bool max_mu_condition(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c){
    return mu < MAX_MU;
}

inline bool max_lambda_condition(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c){
    for(int i = 0; i < M; i++){
        if(lambda[i] >= MAX_LAMBDA){
            return false;
        }
    }
    return true;
}


/*--------------------------------------- MU UPDATING FUNCTIONS ----------------------------------- */

inline mu_Type update_mu_exp(const mu_Type mu, const mu_Type rho){
    return mu * rho;
}

inline mu_Type update_mu_lin(const mu_Type mu, const mu_Type rho){
    return mu + rho;
}


/*--------------------------------------- PRINTING FUNCTIONS -------------------------------------- */

void print_Q(const Q_Type* Q, const dim_Type N);
void print_A(const A_Type* A, const dim_Type M, const dim_Type N);
void print_b(const b_Type* b, const dim_Type M);


/*--------------------------------------- SAVE ON FILE FUNCTIONS ---------------------------------- */

void finalize(test_results mean_results);
void finalize(std::vector<test_results> results);
void print_file_stdout(FILE *file, const char *format, ...);




/* #############################################################################################################################################*/ 


//NB: viene memorizzata solo la diagonale, pertanto Q è di lunghezza N
void fill_Q_diag_lin(Q_Type* Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbund){
    Q_Type RAND_MAX_ = (Q_Type)RAND_MAX;

    std::random_device rd;
    std::mt19937 g(rd());

    for(dim_Type i = 0; i < N; i++){
        Q[i] = lowerbound + (upperbund-lowerbound)*((Q_Type)g()/RAND_MAX_);
    }
}

//NB: viene memorizzata solo la diagonale, pertanto Q è di lunghezza N
void fill_Q_id_lin(Q_Type* Q, const dim_Type N, const Q_Type not_used_1, const Q_Type not_used_2){
    for(dim_Type i = 0; i < N; i++){
            Q[i] = 1;
    }
    printf("WARNING: you're using fill_Q_id_lin, which is quite useless since Q is the identity matrix\n");
}

//NB: non vengono memorizzati gli zeri della matrice triangolare inferiore
void fill_Q_upper_trianular_lin(Q_Type *Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbound){
    Q_Type RAND_MAX_ = (Q_Type)RAND_MAX;
    const unsigned int Q_len = N*(N+1)/2;

    std::random_device rd;
    std::mt19937 g(rd());

    for(dim_Type i = 0; i < Q_len; i++){
        Q[i] = lowerbound + (upperbound-lowerbound)*((Q_Type)g()/RAND_MAX_);
    }
}

void fill_A_neg_binary_lin(A_Type*  A, const dim_Type M, const dim_Type N, const float one_probability, const b_Type b){
    
    A_Type aux_vec[(int)(M * (N - b))] = {(A_Type)0};
    
    for(dim_Type i = 0; i < M; i++){
        for(dim_Type j = 0; j < b; j++){
            A[i+j*M] = -1;
        }
    }

    const unsigned int n_missing_ones = (int)(M * N * one_probability) - M * b;

    for(int i = 0; i < n_missing_ones; i++){
        aux_vec[i] = -1;
    }

    std::random_device rd;
    std::mt19937 g(rd());

    unsigned int aux_vec_len = M * (N - b); 
    std::shuffle(aux_vec, aux_vec + aux_vec_len, g);

    unsigned int c = 0;
    for(dim_Type i = 0; i < M; i++){
        for(dim_Type j = b; j < N; j++){
            A[i+j*M] = aux_vec[c++];
        }
    }
}


void fill_b_vector_lin(b_Type* b, const dim_Type M, const b_Type b_val){
    for(dim_Type i = 0; i < M; i++){
        b[i] = b_val;
    }
}

void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda){
    for(dim_Type i = 0; i < M; i++){
        lambda[i] = initial_lambda;
    }
}

void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda, lambda_Type noise_amplitude){
    const lambda_Type RAND_MAX_ = (lambda_Type)RAND_MAX;
    for(dim_Type i = 0; i < M; i++){
        lambda[i] = initial_lambda + noise_amplitude * (lambda_Type)rand() / RAND_MAX_;
    }
}


void compute_Q_plus_AT_A_upper_triangular_lin(const Q_Type* __restrict__ Q, A_Type* __restrict__ A, A_Type* __restrict__ Q_plus_AT_A, const dim_Type M, const dim_Type N){
    if(!Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                dim_Type triang_idx = triang_index(i,j,N);
                Q_plus_AT_A[triang_idx] = 0;
                for(dim_Type k = 0; k < M; k++){
                    Q_plus_AT_A[triang_idx] += A[k+i*M] * A[k+j*M];
                }
                if(i != j){
                    Q_plus_AT_A[triang_idx] *= 2;
                }
                Q_plus_AT_A[triang_idx] += Q[triang_idx]; 
            }
        }
    } else if(!Q_ID){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                dim_Type triang_idx = triang_index(i,j,N);
                Q_plus_AT_A[triang_idx] = 0;
                for(dim_Type k = 0; k < M; k++){
                    Q_plus_AT_A[triang_idx] += A[k+i*M] * A[k+j*M];
                }
                if(i != j){
                    Q_plus_AT_A[triang_idx] *= 2;
                } 
            }
            Q_plus_AT_A[triang_index(i,i,N)] += Q[i];
        }
    } else {
       for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                dim_Type triang_idx = triang_index(i,j,N);
                Q_plus_AT_A[triang_idx] = 0;
                for(dim_Type k = 0; k < M; k++){
                    Q_plus_AT_A[triang_idx] += A[k+i*M] * A[k+j*M];
                }
                if(i != j){
                    Q_plus_AT_A[triang_idx] *= 2;
                } 
            }
            Q_plus_AT_A[triang_index(i,i,N)] += 1;
        }
    }
}

void print_Q(const Q_Type* Q, const dim_Type N){
    printf("Q =\n");
    if(Q_ID){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = 0; j < N; j++){
                if(i != j)
                    printf("0 ");
                else 
                    printf("%.0f ", Q[i]);
            }
            printf("\n");
        }
        printf("\n");
    } else if(Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = 0; j < N; j++){
                if(i != j)
                    printf("0  ");
                else 
                    printf("%.1f ", Q[i]);
            }
            printf("\n");
        }
        printf("\n");
    } else {
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = 0; j < N; j++){
                printf("%.1f ", Q[i*N+j]);
            }
            printf("\n");
        }
    }
}

void print_A(const A_Type* A, const dim_Type M, const dim_Type N){
    printf("A =\n");
    for(dim_Type i = 0; i < M; i++){
        for(dim_Type j = 0; j < N; j++){
            printf("%3.0f ", A[i+j*M]);
        }
        printf("\n");
    }
}

void print_b(const b_Type* b, const dim_Type M){
    printf("b^T = [");
    for(dim_Type i = 0; i < M; i++){
        printf("%.0f ", b[i]);
    }
    printf("]\n");
}


Q_Type compute_xQx(const Q_Type* __restrict__ Q, const bool* __restrict__ x, dim_Type N){
    Q_Type res = 0;
    if(Q_ID){
        for(dim_Type i = 0; i < N; i++){
            res += x[i];
        }
    } else if(Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            res += x[i] * Q[i];
        }
    } else {
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                res += x[i] * x[j] * Q[triang_index(i,j,N)];
            }
        }
    }
    return res;
};

inline Q_Type compute_max(const Q_Type* __restrict__ Q, dim_Type N){
    Q_Type res = 0;
    if(Q_ID){
        return N;
    } else if(Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            res += Q[i];
        }
    } else {
        unsigned int Q_len = N*(N+1)/2;
        for(dim_Type i = 0; i < Q_len; i++){
            res += Q[i];
        }
    }
    return res;

}


int test_at_dimension(  dim_Type N, dim_Type M, int MAXITER, int N_AL_ATTEMPTS, mu_Type initial_mu, lambda_Type initial_lambda,  mu_Type rho, 
                        void (*fill_Q)(Q_Type *Q, const dim_Type N, const Q_Type lowerbound_or_unused, const Q_Type upperbound_or_unused), Q_Type lb_Q, Q_Type ub_Q, 
                        void (*fill_A)(A_Type* A, const dim_Type M, const dim_Type N, const float one_probability_or_unused, const b_Type b_or_unused), float one_prob, 
                        void (*fill_b)(b_Type* b, const dim_Type M, const b_Type b_val_or_unused), b_Type b_val, 
                        std::function<bool(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c)> al_end_condition, 
                        mu_Type (*update_mu)(const mu_Type mu, const mu_Type rho), 
                        test_results* results, bool verbose, bool strong_verbose)
{

    printf("N = %d\tM = %d\n", N, M);
    
    auto start = std::chrono::high_resolution_clock::now();
    const int progressBarWidth = 100;
    srand(time(0));

    // Allocate
    const unsigned int Q_len = Q_DIAG ? N : N*(N+1)/2;
    const unsigned int A_len = M * N;

    Q_Type* Q = new Q_Type[Q_len];
    A_Type* A = new A_Type[M*N];
    b_Type* b = new b_Type[M];
    lambda_Type* lambda = new lambda_Type[M];
    lambda_Type* old_lambda = new lambda_Type[M];
    bool* expected_min_x = new bool[N];
    bool* min_x = new bool[N];
    b_Type* c = new b_Type[M];


    double true_max_val, true_min_val, al_min_val;

    double mu;
    double old_mu;
    double mean_lambda_on_correct_solutions       = 0,    mean_mu_on_correct_solutions      = 0;
    double mean_lambda_on_unfinished_solutions    = 0,    mean_mu_on_unfinished_solutions   = 0;
    double mean_lambda_on_wrong_solutions         = 0,    mean_mu_on_wrong_solutions        = 0;
    double lambda_min_on_correct_solutions        = DBL_MAX,  lambda_max_on_correct_solutions       = DBL_MIN;     
    double lambda_min_on_unfinished_solutions     = DBL_MAX,  lambda_max_on_unfinished_solutions    = DBL_MIN; 
    double lambda_min_on_wrong_solutions          = DBL_MAX,  lambda_max_on_wrong_solutions         = DBL_MIN; 
    double mean_al_attempts_on_correct_solutions     = 0;
    double mean_al_attempts_on_wrong_solutions       = 0;
    double mean_al_attempts_on_unfinished_solutions  = 0;   

    bool correct, unfinished, wrong;

    int correct_counter = 0;
    int unfinished_counter = 0;
    double normalized_error_mean = 0;



    A_Type*     A_gpu; //input
    Q_Type*     Q_gpu; //input
    b_Type*     b_gpu; //input
    
    bool*       x_bin_buffer_gpu; //buffer
    b_Type*     Ax_b_buffer_gpu;  //buffer

    bool*       feasible_gpu; //output /input
    fx_Type*    fx_gpu; // output / input
    
    x_dec_Type* x_min_gpu; //output
    fx_Type*    fx_min_gpu; //output

    fx_Type*    fx_max_gpu; //output

    CHECK(hipMalloc(&A_gpu, A_len * sizeof(A_Type)));
    CHECK(hipMalloc(&Q_gpu, Q_len * sizeof(Q_Type)));
    CHECK(hipMalloc(&b_gpu, M * sizeof(b_Type)));

    CHECK(hipMalloc(&x_bin_buffer_gpu, N * sizeof(bool) * pow(2,N))); //for each thread (thus each x) a buffer of N bools
    CHECK(hipMalloc(&Ax_b_buffer_gpu, M * sizeof(b_Type) * pow(2,N))); //for each thread (thus each x) a buffer of M b_Type

    CHECK(hipMalloc(&feasible_gpu, pow(2,N) * sizeof(bool)));
    CHECK(hipMalloc(&fx_gpu, pow(2,N) * sizeof(fx_Type)));
    
    CHECK(hipMalloc(&x_min_gpu, sizeof(x_dec_Type)));
    CHECK(hipMalloc(&fx_min_gpu, sizeof(fx_Type)));

    CHECK(hipMalloc(&fx_max_gpu, sizeof(fx_Type)););


    for(int iter = 0; iter < MAXITER; iter++) {
        correct = unfinished = wrong = 0;

        fill_Q(Q, N, lb_Q, ub_Q);
        fill_A(A, M, N, one_prob, b_val);
        fill_b(b, M, b_val);

        if(verbose || strong_verbose){
            printf("-------------------------------------------------------------\n");
            //print Q, A, b
            print_Q(Q, N);
            print_A(A, M, N);
            print_b(b, M);
        }

        mu = initial_mu;
        fill_lambda_lin(lambda, M, initial_lambda, 0);

    
        

        CHECK(hipMemcpy(A_gpu, A, A_len * sizeof(A_Type), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(Q_gpu, Q, Q_len * sizeof(Q_Type), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(b_gpu, b, M * sizeof(b_Type), hipMemcpyHostToDevice));



        int n_threads = min(N_THREADS, (int)pow(2,N));
        dim3 threads_per_block(n_threads);
	    dim3 blocks_per_grid(pow(2,N)/n_threads);          

        brute_force<<<blocks_per_grid, threads_per_block>>>(Q_gpu, A_gpu, b_gpu, N, M, x_bin_buffer_gpu, Ax_b_buffer_gpu, feasible_gpu, fx_gpu);
	    CHECK_KERNELCALL();
	    CHECK(hipDeviceSynchronize());

        reduce_argmin_feasible<<<blocks_per_grid, threads_per_block>>>(fx_gpu, feasible_gpu, fx_min_gpu, x_min_gpu);
        CHECK_KERNELCALL();
	    CHECK(hipDeviceSynchronize());


        int true_min_x_dec;
        CHECK(hipMemcpy(&true_min_val, fx_min_gpu, sizeof(double), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&true_min_x_dec, x_min_gpu, sizeof(int), hipMemcpyDeviceToHost));

        for(int i = 0; i < N; i++){
            expected_min_x[i] = (true_min_x_dec >> i) & 1;
        }

        if(strong_verbose){
            printf("Expected minimum found in x = [ ");
            for(int i = 0; i < N; i++){
                printf("%.0f ", expected_min_x[i]);
            }
            printf("] with value %.1f\n", true_min_val);
        }

        true_max_val = compute_max(Q, N);                                          //TO DO: calcolare il vero massimo

        /*//NB: im skipping the problem if there is no feasible solution. It would be interesting to check if AL realize it.
        if(!find_x_min_brute_force(Q, N, A, M, b, expected_min_x, &true_max_val, &true_min_val, strong_verbose)){
            iter--;
            continue;
        }*/

        int i = 0;
        bool ok;
        bool al_condition;
        
        Q_Type Q_plus_AT_A[N*(N+1)/2];
        compute_Q_plus_AT_A_upper_triangular_lin(Q, A, Q_plus_AT_A, M, N);

        do{

            if(strong_verbose){
                printf("AL attempt %d\tmu = %.5f\tlambda^T = [ ", i, mu);
            }
            
            ok = true;

            printf("DOVRESTI PRIMA SCRIVERE IL KERNEL PER AL. Ti faccio un iterazione di test a vuoto\n");
            
            if(strong_verbose){
                for(int i = 0; i < M; i++){
                    printf("%.5f ", lambda[i]);
                }
                printf("]\tc_x_opt^T = [ ");
                for(int i = 0; i < M; i++){
                    printf("%.5f ", c[i]);
                }
                printf("]\tx_opt = [ ");
                for(int i = 0; i < N; i++){
                    printf("%.0f ", min_x[i]);
                }
                printf("]\tmin_val = %.1f\n", al_min_val);
            }

            for(dim_Type j = 0; j < M; j++){
                old_lambda[j] = lambda[j];
            }
            old_mu = mu;
            

            for(dim_Type j = 0; j < M; j++){
                if(c[j] > 0){
                    lambda[j] = lambda[j] + mu * c[j];               //ORIGINALEEEEEE
                    //lambda[i][0] = lambda[i][0] + rho * c[i][0];

                    ok = false;
                }
            }

            i++;

            mu = update_mu(mu, rho);

            al_condition = al_end_condition(i, N_AL_ATTEMPTS, N, M, lambda, mu, c);

            if(i == 2) ok = true;           //TO DO: da togliere!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        } while (!ok && al_condition);


        Q_Type current_xQx = compute_xQx(Q, min_x, N);
        correct = al_condition && ok && current_xQx == true_min_val;
        unfinished = !al_condition;
        if(correct && unfinished){
            printf("ERROR: the same problem is both correct and unfinished\n");
            return 0;
        } else if(!correct && !unfinished){
            wrong = true;
        }

        if(correct){                      //AL has chosen the right minimum (consciously)
            correct_counter++; 
            if(strong_verbose)
                printf("PROBLEM SOLVED CORRECTLY\n");
        } 
        else if(unfinished){                //AL has reached the termination condition without finding a feasible minimum 
            if(strong_verbose)
                printf("PROBLEM NOT SOLVED\n");                            
            unfinished_counter++;
        }
        else if(wrong){                     //AL has chosen the wrong minimum
            if(strong_verbose)
                printf("PROBLEM SOLVED WRONGLY\n");  
            normalized_error_mean += true_max_val-true_min_val != 0 ? (current_xQx - true_min_val) / (true_max_val-true_min_val) : 1;
            //It DOESN'T make sesnse that the error is negative. true_min_val is the minimum feasible value of the function, if AL exits the loop beleiving that a lower minimum (that could exists) fulfils the constraints, there is a problem while checking c(x)
            if(normalized_error_mean < 0){
                printf("ERROR!\ntrue max val : %.1f\t true min val: %.1f\t xQx: %.1f\n", true_max_val, true_min_val, current_xQx);
                print_Q(Q, N);
                print_A(A, M, N);
                print_b(b, M);
                printf("c = \n");
                for(int i = 0; i < M; i++){
                    printf("%.1f ", c[i]);
                }
                printf("\n");
                return 0;
            }
        } else {
            printf("ERROR: something went wrong\n");
            return 0;
        }
       
        // Print progress bar
        if(!verbose && !strong_verbose){
            printf("[");
            int pos = progressBarWidth * (iter+1) / MAXITER;
            for (int j = 0; j < progressBarWidth; ++j) {
                if (j < pos) printf("=");
                else if (j == pos) printf(">");
                else printf(" ");
            }
            printf("] %d %%\r", int((iter+1) * 100.0 / MAXITER));
            fflush(stdout);

        } else {
            if(ok){
                printf("Problem solved in %d iterations\n", i);
            } else{
                printf("Problem not solved in %d iterations\n", i);
            }

            if(!strong_verbose){
                printf("c_x^T =\t\t[\t");
                for(int i = 0; i < M; i++){
                    printf("%.1f\t", c[i]);
                }
                printf("]\nlambda^T =\t[\t");
                for(int i = 0; i < M; i++){
                    printf("%.1f\t", lambda[i]);
                }
                printf("]\nmu =\t\t%.1f\n\n", mu);
            }
        }


        if(correct){
            mean_al_attempts_on_correct_solutions += i;
            mean_mu_on_correct_solutions += old_mu;
            for(int j = 0; j < M; j++){
                mean_lambda_on_correct_solutions += old_lambda[j]/M;
                if(lambda[j] < lambda_min_on_correct_solutions)
                    lambda_min_on_correct_solutions = lambda[j];
                if(lambda[j] > lambda_max_on_correct_solutions)
                    lambda_max_on_correct_solutions = lambda[j];
            }
        }
        else if(unfinished){
            mean_al_attempts_on_unfinished_solutions += i;
            mean_mu_on_unfinished_solutions += old_mu;
            for(int j = 0; j < M; j++){
                mean_lambda_on_unfinished_solutions += old_lambda[j]/M;
                if(lambda[j] < lambda_min_on_unfinished_solutions)
                    lambda_min_on_unfinished_solutions = lambda[j];
                if(lambda[j] > lambda_max_on_unfinished_solutions)
                    lambda_max_on_unfinished_solutions = lambda[j];
            }
        }
        else if(wrong){
            mean_al_attempts_on_wrong_solutions += i;
            mean_mu_on_wrong_solutions += old_mu;
            for(int j = 0; j < M; j++){
                mean_lambda_on_wrong_solutions += old_lambda[j]/M;
                if(lambda[j] < lambda_min_on_wrong_solutions)
                    lambda_min_on_wrong_solutions = lambda[j];
                if(lambda[j] > lambda_max_on_wrong_solutions)
                    lambda_max_on_wrong_solutions = lambda[j];
            }
        }
        

    }


    mean_lambda_on_correct_solutions = correct_counter != 0 ? mean_lambda_on_correct_solutions / correct_counter : 0;
    mean_mu_on_correct_solutions = correct_counter != 0 ? mean_mu_on_correct_solutions / correct_counter : 0;
    mean_al_attempts_on_correct_solutions = correct_counter != 0 ? mean_al_attempts_on_correct_solutions / correct_counter : 0;

    mean_lambda_on_unfinished_solutions = unfinished_counter != 0 ? mean_lambda_on_unfinished_solutions / unfinished_counter : 0;
    mean_mu_on_unfinished_solutions = unfinished_counter != 0 ? mean_mu_on_unfinished_solutions / unfinished_counter : 0;
    mean_al_attempts_on_unfinished_solutions = unfinished_counter != 0 ? mean_al_attempts_on_unfinished_solutions / unfinished_counter : 0;

    mean_lambda_on_wrong_solutions = MAXITER - correct_counter - unfinished_counter != 0 ? mean_lambda_on_wrong_solutions / (MAXITER - correct_counter - unfinished_counter) : 0;
    mean_mu_on_wrong_solutions = MAXITER - correct_counter - unfinished_counter != 0 ? mean_mu_on_wrong_solutions / (MAXITER - correct_counter - unfinished_counter) : 0;
    mean_al_attempts_on_wrong_solutions = MAXITER - correct_counter - unfinished_counter != 0 ? mean_al_attempts_on_wrong_solutions / (MAXITER - correct_counter - unfinished_counter) : 0;


    normalized_error_mean = MAXITER - correct_counter != 0 ? normalized_error_mean / (MAXITER - correct_counter) : 0;

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    
    results->N = N;
    results->M = M;
    results->mean_al_attempts_on_correct_solutions = mean_al_attempts_on_correct_solutions;
    results->mean_al_attempts_on_wrong_solutions = mean_al_attempts_on_wrong_solutions;
    results->mean_al_attempts_on_unfinished_solutions = mean_al_attempts_on_unfinished_solutions;
    results->correct_ratio = (double)correct_counter/MAXITER;
    results->unfinished_ratio = (double)unfinished_counter/MAXITER;
    results->normalized_error_mean = normalized_error_mean;
    results->mean_lambda_on_correct_solutions = mean_lambda_on_correct_solutions;
    results->mean_lambda_on_unfinished_solutions = mean_lambda_on_unfinished_solutions;
    results->mean_lambda_on_wrong_solutions = mean_lambda_on_wrong_solutions;
    results->lambda_min_on_correct_solutions = lambda_min_on_correct_solutions;
    results->lambda_min_on_unfinished_solutions = lambda_min_on_unfinished_solutions;
    results->lambda_min_on_wrong_solutions = lambda_min_on_wrong_solutions;
    results->lambda_max_on_correct_solutions = lambda_max_on_correct_solutions;
    results->lambda_max_on_unfinished_solutions = lambda_max_on_unfinished_solutions;
    results->lambda_max_on_wrong_solutions = lambda_max_on_wrong_solutions;
    results->mean_mu_on_correct_solutions = mean_mu_on_correct_solutions;
    results->mean_mu_on_unfinished_solutions = mean_mu_on_unfinished_solutions;
    results->mean_mu_on_wrong_solutions = mean_mu_on_wrong_solutions;
    results->duration = elapsed.count();

    //Free GPU memory
    CHECK(hipFree(A_gpu));
    CHECK(hipFree(Q_gpu));
    CHECK(hipFree(b_gpu));
    
    CHECK(hipFree(x_bin_buffer_gpu));
    CHECK(hipFree(Ax_b_buffer_gpu));

    CHECK(hipFree(feasible_gpu));
    CHECK(hipFree(fx_gpu));
    
    CHECK(hipFree(x_min_gpu));
    CHECK(hipFree(fx_min_gpu));

    // Deallocate
    delete[] Q;
    delete[] A;
    delete[] b;
    delete[] lambda;
    delete[] old_lambda;
    delete[] expected_min_x;
    delete[] min_x;
    delete[] c;

    return 1;
}

void finalize(std::vector<test_results> results){
    std::time_t t = std::time(nullptr);
    char mbstr[100];
    std::strftime(mbstr, sizeof(mbstr), "%Y%m%d_%H%M%S", std::localtime(&t));

    std::stringstream filename;
    filename << results_path << "/results_" << mbstr;
    if(strlen(name_suffix) > 0){
        filename << "__" << name_suffix;
    }
    filename << ".csv";

    FILE* file = fopen(filename.str().c_str(), "w");
    fprintf(file, "N,M,correct_ratio,unfinished_ratio,normalized_error_mean,mean_al_attempts_on_correct_solutions,mean_al_attempts_on_wrong_solutions,mean_al_attempts_on_unfinished_solutions,mean_lambda_on_correct_solutions,mean_lambda_on_unfinished_solutions,mean_lambda_on_wrong_solutions,mean_mu_on_correct_solutions,mean_mu_on_unfinished_solutions,mean_mu_on_wrong_solutions,lambda_min_on_correct_solutions,lambda_min_on_unfinished_solutions,lambda_min_on_wrong_solutions,lambda_max_on_correct_solutions,lambda_max_on_unfinished_solutions,lambda_max_on_wrong_solutions,duration\n");
    for(int i = 0; i < results.size(); i++){
        fprintf(file, "%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f%s", 
            results[i].N,
            results[i].M,
            results[i].correct_ratio,
            results[i].unfinished_ratio,
            results[i].normalized_error_mean,
            results[i].mean_al_attempts_on_correct_solutions,
            results[i].mean_al_attempts_on_wrong_solutions,
            results[i].mean_al_attempts_on_unfinished_solutions,
            results[i].mean_lambda_on_correct_solutions,
            results[i].mean_lambda_on_unfinished_solutions,
            results[i].mean_lambda_on_wrong_solutions,
            results[i].mean_mu_on_correct_solutions,
            results[i].mean_mu_on_unfinished_solutions,
            results[i].mean_mu_on_wrong_solutions,
            results[i].lambda_min_on_correct_solutions,
            results[i].lambda_min_on_unfinished_solutions,
            results[i].lambda_min_on_wrong_solutions,
            results[i].lambda_max_on_correct_solutions,
            results[i].lambda_max_on_unfinished_solutions,
            results[i].lambda_max_on_wrong_solutions,
            results[i].duration,
            i < results.size()-1 ? "\n" : ""
            );
        }

    fclose(file);
}

void finalize(test_results mean_results){
    std::time_t t = std::time(nullptr);
    char mbstr[100];
    std::strftime(mbstr, sizeof(mbstr), "%Y%m%d_%H%M%S", std::localtime(&t));

    std::stringstream filename;
    filename << results_path << "mean_results_" << mbstr;
    if(strlen(name_suffix) > 0){
        filename << "__" << name_suffix;
    }
    filename << ".csv";

    FILE* file = fopen(filename.str().c_str(), "w");
    fprintf(file, "N,M,correct_ratio,unfinished_ratio,normalized_error_mean,mean_al_attempts_on_correct_solutions,mean_al_attempts_on_wrong_solutions,mean_al_attempts_on_unfinished_solutions,mean_lambda_on_correct_solutions,mean_lambda_on_unfinished_solutions,mean_lambda_on_wrong_solutions,mean_mu_on_correct_solutions,mean_mu_on_unfinished_solutions,mean_mu_on_wrong_solutions,lambda_min_on_correct_solutions,lambda_min_on_unfinished_solutions,lambda_min_on_wrong_solutions,lambda_max_on_correct_solutions,lambda_max_on_unfinished_solutions,lambda_max_on_wrong_solutions,duration\n");
    fprintf(file, "%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f", 
        mean_results.N,
        mean_results.M,
        mean_results.correct_ratio,
        mean_results.unfinished_ratio,
        mean_results.normalized_error_mean,
        mean_results.mean_al_attempts_on_correct_solutions,
        mean_results.mean_al_attempts_on_wrong_solutions,
        mean_results.mean_al_attempts_on_unfinished_solutions,
        mean_results.mean_lambda_on_correct_solutions,
        mean_results.mean_lambda_on_unfinished_solutions,
        mean_results.mean_lambda_on_wrong_solutions,
        mean_results.mean_mu_on_correct_solutions,
        mean_results.mean_mu_on_unfinished_solutions,
        mean_results.mean_mu_on_wrong_solutions,
        mean_results.lambda_min_on_correct_solutions,
        mean_results.lambda_min_on_unfinished_solutions,
        mean_results.lambda_min_on_wrong_solutions,
        mean_results.lambda_max_on_correct_solutions,
        mean_results.lambda_max_on_unfinished_solutions,
        mean_results.lambda_max_on_wrong_solutions,
        mean_results.duration       
        );
    
    fclose(file);
}

void print_file_stdout(FILE *file, const char *format, ...) {
    va_list args;
    va_start(args, format);
    vprintf(format, args); // Print to stdout
    va_end(args);

    va_start(args, format);
    vfprintf(file, format, args); // Print to file
    va_end(args);
}