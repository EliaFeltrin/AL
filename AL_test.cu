#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <time.h>
#include <chrono>
#include <ctime>
#include <sstream>
#include <cstdlib>
#include <unistd.h>
#include <float.h>
#include <functional>
#include <cstring>
#include <cstdarg>
#include <random>
#include <unordered_set>
#include <hip/hip_runtime.h>


#include "types.h"
#include "kernels.cu"

extern double MAX_MU;
extern double MAX_LAMBDA;

extern char name_suffix[20];  
extern char results_path[100];

enum stop_conditions_names {max_Al_attempts, max_mu, max_lambda, stop_conditions_end};
enum fill_distributions {uniform, MMF, PCR, PCRL, fill_distributions_end};

extern bool Q_DIAG;
extern bool Q_ID;
extern bool PCR_PROBLEM;

#define CHECK(call)                                                                         \
	{                                                                                       \
		const hipError_t err = call;                                                       \
		if (err != hipSuccess) {                                                           \
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
			exit(EXIT_FAILURE);                                                             \
		}                                                                                   \
	}

#define CHECK_KERNELCALL()                                                                  \
	{                                                                                       \
		const hipError_t err = hipGetLastError();                                         \
		if (err != hipSuccess) {                                                           \
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
			exit(EXIT_FAILURE);                                                             \
		}                                                                                   \
	} 

struct test_results{
    dim_Type N;
    dim_Type M;
    double correct_ratio;
    double unfinished_ratio;
    double normalized_error_mean;
    float mean_al_attempts_on_correct_solutions;
    float mean_al_attempts_on_wrong_solutions;
    float mean_al_attempts_on_unfinished_solutions;
    lambda_Type mean_lambda_on_correct_solutions;
    lambda_Type mean_lambda_on_unfinished_solutions;
    lambda_Type mean_lambda_on_wrong_solutions;
    mu_Type mean_mu_on_correct_solutions;
    mu_Type mean_mu_on_unfinished_solutions;
    mu_Type mean_mu_on_wrong_solutions;
    lambda_Type lambda_min_on_correct_solutions;
    lambda_Type lambda_min_on_unfinished_solutions;
    lambda_Type lambda_min_on_wrong_solutions;
    lambda_Type lambda_max_on_correct_solutions;
    lambda_Type lambda_max_on_unfinished_solutions;
    lambda_Type lambda_max_on_wrong_solutions;
    double duration;
};


void fill_Q_id_lin(Q_Type*  Q, const dim_Type N, const Q_Type not_used_1, const Q_Type not_used_2);
void fill_Q_diag_lin(Q_Type*  Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbund);
void fill_Q_upper_trianular_lin(Q_Type *Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbound);
void fill_Q_manual_lin(Q_Type*  Q, const dim_Type N, const Q_Type unused_1, const Q_Type unused_2) {printf("not yet implemented"); exit(0);}

void fill_A_neg_binary_lin(A_Type*  A, const dim_Type M, const dim_Type N, const float one_probability, const b_Type b);
void fill_A_manual_lin(A_Type*  A, const dim_Type M, const dim_Type N, const float unused_1, const b_Type unused_2) {printf("not yet implemented"); exit(0);}

void fill_b_vector_lin(b_Type* b, const dim_Type M, const b_Type b_val);
void fill_b_manual_lin(b_Type* b, const dim_Type M, const b_Type unused) {printf("not yet implemented"); exit(0);}

void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda);
void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda, lambda_Type noise_amplitude);

int test_at_dimension(  dim_Type N, dim_Type M, int MAXITER, int N_AL_ATTEMPTS, mu_Type initial_mu, lambda_Type initial_lambda,  mu_Type rho, 
                        void (*fill_Q)(Q_Type *Q, const dim_Type N, const Q_Type lowerbound_or_unused, const Q_Type upperbound_or_unused), Q_Type lb_Q, Q_Type ub_Q, 
                        void (*fill_A)(A_Type* A, const dim_Type M, const dim_Type N, const float one_probability_or_unused, const b_Type b_or_unused), float one_prob, 
                        void (*fill_b)(b_Type* b, const dim_Type M, const b_Type b_val_or_unused), b_Type b_val, 
                        std::function<bool(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c)> al_end_condition, 
                        mu_Type (*update_mu)(const mu_Type mu, const mu_Type rho), 
                        test_results* results, bool verbose, bool strong_verbose);

void print_Q(const Q_Type* Q, const dim_Type N);
void print_A(const A_Type* A, const dim_Type M, const dim_Type N);
void print_b(const b_Type* b, const dim_Type M);


Q_Type compute_xQx(const Q_Type* __restrict__ Q, const bool* __restrict__ x, dim_Type N);


inline mu_Type update_mu_exp(const mu_Type mu, const mu_Type rho){
    return mu * rho;
}

inline mu_Type update_mu_lin(const mu_Type mu, const mu_Type rho){
    return mu + rho;
}


void compute_Q_plus_AT_A_upper_triangular_lin(const Q_Type* __restrict__ Q, A_Type* __restrict__ A, A_Type* __restrict__ Q_plus_AT_A, const dim_Type M, const dim_Type N);

inline dim_Type triang_index(dim_Type i, dim_Type j, dim_Type N){
    return i * (N - 0.5) - i*i/2 + j;
}

void finalize(test_results mean_results);
void finalize(std::vector<test_results> results);
void print_file_stdout(FILE *file, const char *format, ...);














//NB: viene memorizzata solo la diagonale, pertanto Q è di lunghezza N
void fill_Q_diag_lin(Q_Type* Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbund){
    Q_Type RAND_MAX_ = (Q_Type)RAND_MAX;
    for(dim_Type i = 0; i < N; i++){
        Q[i*N+i] = lowerbound + (upperbund-lowerbound)*((Q_Type)rand()/RAND_MAX_);
    }
}

//NB: viene memorizzata solo la diagonale, pertanto Q è di lunghezza N
void fill_Q_id_lin(Q_Type* Q, const dim_Type N, const Q_Type not_used_1, const Q_Type not_used_2){
    for(dim_Type i = 0; i < N; i++){
            Q[i] = 1;
    }
    printf("WARNING: you're using fill_Q_id_lin, which is quite useless since Q is the identity matrix");
}

//NB: non vengono memorizzati gli zeri della matrice triangolare inferiore
void fill_Q_upper_trianular_lin(Q_Type *Q, const dim_Type N, const Q_Type lowerbound, const Q_Type upperbound){
    Q_Type RAND_MAX_ = (Q_Type)RAND_MAX;
    for(dim_Type i = 0; i < N; i++){
        for(dim_Type j = i; j < N; j++){
            Q[i*N+j] = lowerbound + (upperbound-lowerbound)*((Q_Type)rand()/RAND_MAX_);
        }
    }
}

void fill_A_neg_binary_lin(A_Type*  A, const dim_Type M, const dim_Type N, const float one_probability, const b_Type b){
    
    A_Type aux_vec[(int)(M * (N - b))] = {(A_Type)0};
    
    for(dim_Type i = 0; i < M; i++){
        for(dim_Type j = 0; j < b; j++){
            A[i+j*M] = -1;
        }
    }

    const unsigned int n_missing_ones = (int)(M * N * one_probability) - M * b;

    for(int i = 0; i < n_missing_ones; i++){
        aux_vec[i] = -1;
    }

    std::random_device rd;
    std::mt19937 g(rd());

    unsigned int aux_vec_len = M * (N - b); 
    std::shuffle(aux_vec, aux_vec + aux_vec_len, g);

    unsigned int c = 0;
    for(dim_Type i = 0; i < M; i++){
        for(dim_Type j = b; j < N; j++){
            A[i+j*M] = aux_vec[c++];
        }
    }
}


void fill_b_vector_lin(b_Type* b, const dim_Type M, const b_Type b_val){
    for(dim_Type i = 0; i < M; i++){
        b[i] = b_val;
    }
}

void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda){
    for(dim_Type i = 0; i < M; i++){
        lambda[i] = initial_lambda;
    }
}

void fill_lambda_lin(lambda_Type* lambda, const dim_Type M, lambda_Type initial_lambda, lambda_Type noise_amplitude){
    const lambda_Type RAND_MAX_ = (lambda_Type)RAND_MAX;
    for(dim_Type i = 0; i < M; i++){
        lambda[i] = initial_lambda + noise_amplitude * (lambda_Type)rand() / RAND_MAX_;
    }
}


void compute_Q_plus_AT_A_upper_triangular_lin(const Q_Type* __restrict__ Q, A_Type* __restrict__ A, A_Type* __restrict__ Q_plus_AT_A, const dim_Type M, const dim_Type N){
    if(!Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                dim_Type triang_idx = triang_index(i,j,N);
                Q_plus_AT_A[triang_idx] = 0;
                for(dim_Type k = 0; k < M; k++){
                    Q_plus_AT_A[triang_idx] += A[k+i*M] * A[k+j*M];
                }
                if(i != j){
                    Q_plus_AT_A[triang_idx] *= 2;
                }
                Q_plus_AT_A[triang_idx] += Q[triang_idx]; 
            }
        }
    } else if(!Q_ID){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                dim_Type triang_idx = triang_index(i,j,N);
                Q_plus_AT_A[triang_idx] = 0;
                for(dim_Type k = 0; k < M; k++){
                    Q_plus_AT_A[triang_idx] += A[k+i*M] * A[k+j*M];
                }
                if(i != j){
                    Q_plus_AT_A[triang_idx] *= 2;
                } 
            }
            Q_plus_AT_A[triang_index(i,i,N)] += Q[i];
        }
    } else {
       for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                dim_Type triang_idx = triang_index(i,j,N);
                Q_plus_AT_A[triang_idx] = 0;
                for(dim_Type k = 0; k < M; k++){
                    Q_plus_AT_A[triang_idx] += A[k+i*M] * A[k+j*M];
                }
                if(i != j){
                    Q_plus_AT_A[triang_idx] *= 2;
                } 
            }
            Q_plus_AT_A[triang_index(i,i,N)] += 1;
        }
    }
}

void print_Q(const Q_Type* Q, const dim_Type N){
    printf("Q =\n");
    if(Q_ID){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = 0; j < N; j++){
                if(i != j)
                    printf("0 ");
                else 
                    printf("%.0f ", Q[i]);
            }
        }
        printf("\n");
    } else if(Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = 0; j < N; j++){
                if(i != j)
                    printf("0  ");
                else 
                    printf("%.1f ", Q[i]);
            }
        }
        printf("\n");
    } else {
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = 0; j < N; j++){
                printf("%.1f ", Q[i*N+j]);
            }
            printf("\n");
        }
    }
}

void print_A(const A_Type* A, const dim_Type M, const dim_Type N){
    printf("A =\n");
    for(dim_Type i = 0; i < M; i++){
        for(dim_Type j = 0; j < N; j++){
            printf("%3.0f ", A[i+j*M]);
        }
        printf("\n");
    }
}

void print_b(const b_Type* b, const dim_Type M){
    printf("b^T = [");
    for(dim_Type i = 0; i < M; i++){
        printf("%.0f ", b[i]);
    }
    printf("]\n");
}


Q_Type compute_xQx(const Q_Type* __restrict__ Q, const bool* __restrict__ x, dim_Type N){
    Q_Type res = 0;
    if(Q_ID){
        for(dim_Type i = 0; i < N; i++){
            res += x[i];
        }
    } else if(Q_DIAG){
        for(dim_Type i = 0; i < N; i++){
            res += x[i] * Q[i];
        }
    } else {
        for(dim_Type i = 0; i < N; i++){
            for(dim_Type j = i; j < N; j++){
                res += x[i] * x[j] * Q[triang_index(i,j,N)];
            }
        }
    }
    return res;
};


int test_at_dimension(  dim_Type N, dim_Type M, int MAXITER, int N_AL_ATTEMPTS, mu_Type initial_mu, lambda_Type initial_lambda,  mu_Type rho, 
                        void (*fill_Q)(Q_Type *Q, const dim_Type N, const Q_Type lowerbound_or_unused, const Q_Type upperbound_or_unused), Q_Type lb_Q, Q_Type ub_Q, 
                        void (*fill_A)(A_Type* A, const dim_Type M, const dim_Type N, const float one_probability_or_unused, const b_Type b_or_unused), float one_prob, 
                        void (*fill_b)(b_Type* b, const dim_Type M, const b_Type b_val_or_unused), b_Type b_val, 
                        std::function<bool(const int i, const int N_AL_ATTEMPTS, const dim_Type N, const dim_Type M, const lambda_Type* __restrict__ lambda, const mu_Type mu, const b_Type* __restrict__ c)> al_end_condition, 
                        mu_Type (*update_mu)(const mu_Type mu, const mu_Type rho), 
                        test_results* results, bool verbose, bool strong_verbose)
{

    printf("N = %d\tM = %d\n", N, M);
    
    auto start = std::chrono::high_resolution_clock::now();
    const int progressBarWidth = 100;
    srand(time(0));

    // Allocate
    Q_Type* Q = new Q_Type[N*(N+1)/2];
    A_Type* A = new A_Type[M*N];
    b_Type* b = new b_Type[M];
    lambda_Type* lambda = new lambda_Type[M];
    lambda_Type* old_lambda = new lambda_Type[M];
    bool* expected_min_x = new bool[N];
    bool* min_x = new bool[N];
    b_Type* c = new b_Type[M];


    double true_max_val, true_min_val, al_min_val;

    double mu;
    double old_mu;
    double mean_lambda_on_correct_solutions       = 0,    mean_mu_on_correct_solutions      = 0;
    double mean_lambda_on_unfinished_solutions    = 0,    mean_mu_on_unfinished_solutions   = 0;
    double mean_lambda_on_wrong_solutions         = 0,    mean_mu_on_wrong_solutions        = 0;
    double lambda_min_on_correct_solutions        = DBL_MAX,  lambda_max_on_correct_solutions       = DBL_MIN;     
    double lambda_min_on_unfinished_solutions     = DBL_MAX,  lambda_max_on_unfinished_solutions    = DBL_MIN; 
    double lambda_min_on_wrong_solutions          = DBL_MAX,  lambda_max_on_wrong_solutions         = DBL_MIN; 
    double mean_al_attempts_on_correct_solutions     = 0;
    double mean_al_attempts_on_wrong_solutions       = 0;
    double mean_al_attempts_on_unfinished_solutions  = 0;   

    bool correct, unfinished, wrong;

    int correct_counter = 0;
    int unfinished_counter = 0;
    double normalized_error_mean = 0;

    for(int iter = 0; iter < MAXITER; iter++) {
        correct = unfinished = wrong = 0;

        fill_Q(Q, N, lb_Q, ub_Q);
        fill_A(A, M, N, one_prob, b_val);
        fill_b(b, M, b_val);

        if(verbose || strong_verbose){
            printf("-------------------------------------------------------------\n");
            //print Q, A, b
            print_Q(Q, N);
            print_A(A, M, N);
            print_b(b, M);
        }

        mu = initial_mu;
        fill_lambda_lin(lambda, M, initial_lambda, 0);

    
        A_Type*       A_gpu;
        Q_Type*       Q_gpu;
        b_Type*       b_gpu;
        bool*         feasible_gpu;
        fx_Type*      fx_gpu; 
        int*          x_min_gpu;
        fx_Type*       fx_min_gpu;

        CHECK(hipMalloc(&A_gpu, M * N * sizeof(A_Type)));
        CHECK(hipMalloc(&Q_gpu, N * N * sizeof(Q_Type)));
        CHECK(hipMalloc(&b_gpu, M * sizeof(b_Type)));
        CHECK(hipMalloc(&feasible_gpu, pow(2,N) * sizeof(bool)));
        CHECK(hipMalloc(&fx_gpu, pow(2,N) * sizeof(fx_Type)));
        CHECK(hipMalloc(&x_min_gpu, sizeof(int)));
        CHECK(hipMalloc(&fx_min_gpu, sizeof(double)));

        CHECK(hipMemcpy(A_gpu, A, M * N * sizeof(A_Type), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(Q_gpu, Q, N * N * sizeof(Q_Type), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(b_gpu, b, M * sizeof(b_Type), hipMemcpyHostToDevice));


        dim3 threads_per_block(1024);
	    dim3 blocks_per_grid(pow(2,N-10));          ///RICORDATI DI PARAMETRIZZARE QUESTA ROBA PER N DIVERSI

        brute_force<<<blocks_per_grid, threads_per_block>>>(Q_gpu, A_gpu, b_gpu, N, M, feasible_gpu, fx_gpu);
	    CHECK_KERNELCALL();
	    CHECK(hipDeviceSynchronize());

        reduce_argmin_feasible<<<blocks_per_grid, threads_per_block>>>(fx_gpu, feasible_gpu, fx_min_gpu, x_min_gpu);

        CHECK_KERNELCALL();
	    CHECK(hipDeviceSynchronize());

        //TO DO: da fare reduce_max_feasible pere valcolare true_max_val e quindi gli errori

        int true_min_x_dec;
        CHECK(hipMemcpy(&true_min_val, fx_min_gpu, sizeof(double), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(&true_min_x_dec, x_min_gpu, sizeof(int), hipMemcpyDeviceToHost));

        for(int i = 0; i< N; i++){
            expected_min_x[i] = (true_min_x_dec >> i) & 0b1;
        }

        if(strong_verbose){
            printf("Expected minimum found in x = [ ");
            for(int i = 0; i < N; i++){
                printf("%.0f ", expected_min_x[i]);
            }
            printf("] with value %.1f\n", true_min_val);
        }

        true_max_val = 100000;                                          //TO DO: calcolare il vero massimo

        /*//NB: im skipping the problem if there is no feasible solution. It would be interesting to check if AL realize it.
        if(!find_x_min_brute_force(Q, N, A, M, b, expected_min_x, &true_max_val, &true_min_val, strong_verbose)){
            iter--;
            continue;
        }*/

        int i = 0;
        bool ok;
        bool al_condition;
        do{

            if(strong_verbose){
                printf("AL attempt %d\tmu = %.5f\tlambda^T = [ ", i, mu);
            }
            
            ok = true;

            printf("DOVRESTI PRIMA SCRIVERE IL KERNEL PER AL. Ti faccio un iterazione di test a vuoto\n");
            
            if(strong_verbose){
                for(int i = 0; i < M; i++){
                    printf("%.5f ", lambda[i]);
                }
                printf("]\tc_x_opt^T = [ ");
                for(int i = 0; i < M; i++){
                    printf("%.5f ", c[i]);
                }
                printf("]\tx_opt = [ ");
                for(int i = 0; i < N; i++){
                    printf("%.0f ", min_x[i]);
                }
                printf("]\tmin_val = %.1f\n", al_min_val);
            }

            for(dim_Type j = 0; j < M; j++){
                old_lambda[j] = lambda[j];
            }
            old_mu = mu;
            

            for(dim_Type j = 0; j < M; j++){
                if(c[j] > 0){
                    lambda[j] = lambda[j] + mu * c[j];               //ORIGINALEEEEEE
                    //lambda[i][0] = lambda[i][0] + rho * c[i][0];

                    ok = false;
                }
            }

            i++;

            mu = update_mu(mu, rho);

            al_condition = al_end_condition(i, N_AL_ATTEMPTS, N, M, lambda, mu, c);

            if(i == 2) ok = true;           //TO DO: da togliere!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        } while (!ok && al_condition);


        Q_Type current_xQx = compute_xQx(Q, min_x, N);
        correct = al_condition && ok && current_xQx == true_min_val;
        unfinished = !al_condition;
        if(correct && unfinished){
            printf("ERROR: the same problem is both correct and unfinished\n");
            return 0;
        } else if(!correct && !unfinished){
            wrong = true;
        }

        if(correct){                      //AL has chosen the right minimum (consciously)
            correct_counter++; 
            if(strong_verbose)
                printf("PROBLEM SOLVED CORRECTLY\n");
        } 
        else if(unfinished){                //AL has reached the termination condition without finding a feasible minimum 
            if(strong_verbose)
                printf("PROBLEM NOT SOLVED\n");                            
            unfinished_counter++;
        }
        else if(wrong){                     //AL has chosen the wrong minimum
            if(strong_verbose)
                printf("PROBLEM SOLVED WRONGLY\n");  
            normalized_error_mean += true_max_val-true_min_val != 0 ? (current_xQx - true_min_val) / (true_max_val-true_min_val) : 1;
            //It DOESN'T make sesnse that the error is negative. true_min_val is the minimum feasible value of the function, if AL exits the loop beleiving that a lower minimum (that could exists) fulfils the constraints, there is a problem while checking c(x)
            if(normalized_error_mean < 0){
                printf("ERROR!\ntrue max val : %.1f\t true min val: %.1f\t xQx: %.1f\n", true_max_val, true_min_val, current_xQx);
                print_Q(Q, N);
                print_A(A, M, N);
                print_b(b, M);
                printf("c = \n");
                for(int i = 0; i < M; i++){
                    printf("%.1f ", c[i]);
                }
                printf("\n");
                return 0;
            }
        } else {
            printf("ERROR: something went wrong\n");
            return 0;
        }
       
        // Print progress bar
        if(!verbose && !strong_verbose){
            printf("[");
            int pos = progressBarWidth * (iter+1) / MAXITER;
            for (int j = 0; j < progressBarWidth; ++j) {
                if (j < pos) printf("=");
                else if (j == pos) printf(">");
                else printf(" ");
            }
            printf("] %d %%\r", int((iter+1) * 100.0 / MAXITER));
            fflush(stdout);

        } else {
            if(ok){
                printf("Problem solved in %d iterations\n", i);
            } else{
                printf("Problem not solved in %d iterations\n", i);
            }

            if(!strong_verbose){
                printf("c_x^T =\t\t[\t");
                for(int i = 0; i < M; i++){
                    printf("%.1f\t", c[i]);
                }
                printf("]\nlambda^T =\t[\t");
                for(int i = 0; i < M; i++){
                    printf("%.1f\t", lambda[i]);
                }
                printf("]\nmu =\t\t%.1f\n\n", mu);
            }
        }


        if(correct){
            mean_al_attempts_on_correct_solutions += i;
            mean_mu_on_correct_solutions += old_mu;
            for(int j = 0; j < M; j++){
                mean_lambda_on_correct_solutions += old_lambda[j]/M;
                if(lambda[j] < lambda_min_on_correct_solutions)
                    lambda_min_on_correct_solutions = lambda[j];
                if(lambda[j] > lambda_max_on_correct_solutions)
                    lambda_max_on_correct_solutions = lambda[j];
            }
        }
        else if(unfinished){
            mean_al_attempts_on_unfinished_solutions += i;
            mean_mu_on_unfinished_solutions += old_mu;
            for(int j = 0; j < M; j++){
                mean_lambda_on_unfinished_solutions += old_lambda[j]/M;
                if(lambda[j] < lambda_min_on_unfinished_solutions)
                    lambda_min_on_unfinished_solutions = lambda[j];
                if(lambda[j] > lambda_max_on_unfinished_solutions)
                    lambda_max_on_unfinished_solutions = lambda[j];
            }
        }
        else if(wrong){
            mean_al_attempts_on_wrong_solutions += i;
            mean_mu_on_wrong_solutions += old_mu;
            for(int j = 0; j < M; j++){
                mean_lambda_on_wrong_solutions += old_lambda[j]/M;
                if(lambda[j] < lambda_min_on_wrong_solutions)
                    lambda_min_on_wrong_solutions = lambda[j];
                if(lambda[j] > lambda_max_on_wrong_solutions)
                    lambda_max_on_wrong_solutions = lambda[j];
            }
        }
        

    }
    
    mean_lambda_on_correct_solutions = correct_counter != 0 ? mean_lambda_on_correct_solutions / correct_counter : 0;
    mean_mu_on_correct_solutions = correct_counter != 0 ? mean_mu_on_correct_solutions / correct_counter : 0;
    mean_al_attempts_on_correct_solutions = correct_counter != 0 ? mean_al_attempts_on_correct_solutions / correct_counter : 0;

    mean_lambda_on_unfinished_solutions = unfinished_counter != 0 ? mean_lambda_on_unfinished_solutions / unfinished_counter : 0;
    mean_mu_on_unfinished_solutions = unfinished_counter != 0 ? mean_mu_on_unfinished_solutions / unfinished_counter : 0;
    mean_al_attempts_on_unfinished_solutions = unfinished_counter != 0 ? mean_al_attempts_on_unfinished_solutions / unfinished_counter : 0;

    mean_lambda_on_wrong_solutions = MAXITER - correct_counter - unfinished_counter != 0 ? mean_lambda_on_wrong_solutions / (MAXITER - correct_counter - unfinished_counter) : 0;
    mean_mu_on_wrong_solutions = MAXITER - correct_counter - unfinished_counter != 0 ? mean_mu_on_wrong_solutions / (MAXITER - correct_counter - unfinished_counter) : 0;
    mean_al_attempts_on_wrong_solutions = MAXITER - correct_counter - unfinished_counter != 0 ? mean_al_attempts_on_wrong_solutions / (MAXITER - correct_counter - unfinished_counter) : 0;


    normalized_error_mean = MAXITER - correct_counter != 0 ? normalized_error_mean / (MAXITER - correct_counter) : 0;

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    
    results->N = N;
    results->M = M;
    results->mean_al_attempts_on_correct_solutions = mean_al_attempts_on_correct_solutions;
    results->mean_al_attempts_on_wrong_solutions = mean_al_attempts_on_wrong_solutions;
    results->mean_al_attempts_on_unfinished_solutions = mean_al_attempts_on_unfinished_solutions;
    results->correct_ratio = (double)correct_counter/MAXITER;
    results->unfinished_ratio = (double)unfinished_counter/MAXITER;
    results->normalized_error_mean = normalized_error_mean;
    results->mean_lambda_on_correct_solutions = mean_lambda_on_correct_solutions;
    results->mean_lambda_on_unfinished_solutions = mean_lambda_on_unfinished_solutions;
    results->mean_lambda_on_wrong_solutions = mean_lambda_on_wrong_solutions;
    results->lambda_min_on_correct_solutions = lambda_min_on_correct_solutions;
    results->lambda_min_on_unfinished_solutions = lambda_min_on_unfinished_solutions;
    results->lambda_min_on_wrong_solutions = lambda_min_on_wrong_solutions;
    results->lambda_max_on_correct_solutions = lambda_max_on_correct_solutions;
    results->lambda_max_on_unfinished_solutions = lambda_max_on_unfinished_solutions;
    results->lambda_max_on_wrong_solutions = lambda_max_on_wrong_solutions;
    results->mean_mu_on_correct_solutions = mean_mu_on_correct_solutions;
    results->mean_mu_on_unfinished_solutions = mean_mu_on_unfinished_solutions;
    results->mean_mu_on_wrong_solutions = mean_mu_on_wrong_solutions;
    results->duration = elapsed.count();


    // Deallocate
    delete[] Q;
    delete[] A;
    delete[] b;
    delete[] lambda;
    delete[] old_lambda;
    delete[] expected_min_x;
    delete[] min_x;
    delete[] c;

    return 1;
}

void finalize(std::vector<test_results> results){
    std::time_t t = std::time(nullptr);
    char mbstr[100];
    std::strftime(mbstr, sizeof(mbstr), "%Y%m%d_%H%M%S", std::localtime(&t));

    std::stringstream filename;
    filename << results_path << "/results_" << mbstr;
    if(strlen(name_suffix) > 0){
        filename << "__" << name_suffix;
    }
    filename << ".csv";

    FILE* file = fopen(filename.str().c_str(), "w");
    fprintf(file, "N,M,correct_ratio,unfinished_ratio,normalized_error_mean,mean_al_attempts_on_correct_solutions,mean_al_attempts_on_wrong_solutions,mean_al_attempts_on_unfinished_solutions,mean_lambda_on_correct_solutions,mean_lambda_on_unfinished_solutions,mean_lambda_on_wrong_solutions,mean_mu_on_correct_solutions,mean_mu_on_unfinished_solutions,mean_mu_on_wrong_solutions,lambda_min_on_correct_solutions,lambda_min_on_unfinished_solutions,lambda_min_on_wrong_solutions,lambda_max_on_correct_solutions,lambda_max_on_unfinished_solutions,lambda_max_on_wrong_solutions,duration\n");
    for(int i = 0; i < results.size(); i++){
        fprintf(file, "%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f%s", 
            results[i].N,
            results[i].M,
            results[i].correct_ratio,
            results[i].unfinished_ratio,
            results[i].normalized_error_mean,
            results[i].mean_al_attempts_on_correct_solutions,
            results[i].mean_al_attempts_on_wrong_solutions,
            results[i].mean_al_attempts_on_unfinished_solutions,
            results[i].mean_lambda_on_correct_solutions,
            results[i].mean_lambda_on_unfinished_solutions,
            results[i].mean_lambda_on_wrong_solutions,
            results[i].mean_mu_on_correct_solutions,
            results[i].mean_mu_on_unfinished_solutions,
            results[i].mean_mu_on_wrong_solutions,
            results[i].lambda_min_on_correct_solutions,
            results[i].lambda_min_on_unfinished_solutions,
            results[i].lambda_min_on_wrong_solutions,
            results[i].lambda_max_on_correct_solutions,
            results[i].lambda_max_on_unfinished_solutions,
            results[i].lambda_max_on_wrong_solutions,
            results[i].duration,
            i < results.size()-1 ? "\n" : ""
            );
        }

    fclose(file);
}

void finalize(test_results mean_results){
    std::time_t t = std::time(nullptr);
    char mbstr[100];
    std::strftime(mbstr, sizeof(mbstr), "%Y%m%d_%H%M%S", std::localtime(&t));

    std::stringstream filename;
    filename << results_path << "mean_results_" << mbstr;
    if(strlen(name_suffix) > 0){
        filename << "__" << name_suffix;
    }
    filename << ".csv";

    FILE* file = fopen(filename.str().c_str(), "w");
    fprintf(file, "N,M,correct_ratio,unfinished_ratio,normalized_error_mean,mean_al_attempts_on_correct_solutions,mean_al_attempts_on_wrong_solutions,mean_al_attempts_on_unfinished_solutions,mean_lambda_on_correct_solutions,mean_lambda_on_unfinished_solutions,mean_lambda_on_wrong_solutions,mean_mu_on_correct_solutions,mean_mu_on_unfinished_solutions,mean_mu_on_wrong_solutions,lambda_min_on_correct_solutions,lambda_min_on_unfinished_solutions,lambda_min_on_wrong_solutions,lambda_max_on_correct_solutions,lambda_max_on_unfinished_solutions,lambda_max_on_wrong_solutions,duration\n");
    fprintf(file, "%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f", 
        mean_results.N,
        mean_results.M,
        mean_results.correct_ratio,
        mean_results.unfinished_ratio,
        mean_results.normalized_error_mean,
        mean_results.mean_al_attempts_on_correct_solutions,
        mean_results.mean_al_attempts_on_wrong_solutions,
        mean_results.mean_al_attempts_on_unfinished_solutions,
        mean_results.mean_lambda_on_correct_solutions,
        mean_results.mean_lambda_on_unfinished_solutions,
        mean_results.mean_lambda_on_wrong_solutions,
        mean_results.mean_mu_on_correct_solutions,
        mean_results.mean_mu_on_unfinished_solutions,
        mean_results.mean_mu_on_wrong_solutions,
        mean_results.lambda_min_on_correct_solutions,
        mean_results.lambda_min_on_unfinished_solutions,
        mean_results.lambda_min_on_wrong_solutions,
        mean_results.lambda_max_on_correct_solutions,
        mean_results.lambda_max_on_unfinished_solutions,
        mean_results.lambda_max_on_wrong_solutions,
        mean_results.duration       
        );
    
    fclose(file);
}

void print_file_stdout(FILE *file, const char *format, ...) {
    va_list args;
    va_start(args, format);
    vprintf(format, args); // Print to stdout
    va_end(args);

    va_start(args, format);
    vfprintf(file, format, args); // Print to file
    va_end(args);
}